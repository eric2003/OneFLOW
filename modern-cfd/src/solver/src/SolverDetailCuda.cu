
#include <hip/hip_runtime.h>
/*---------------------------------------------------------------------------*\
OneFLOW - LargeScale Multiphysics Scientific Simulation Environment
Copyright (C) 2017-2024 He Xin and the OneFLOW contributors.
-------------------------------------------------------------------------------
License
This file is part of OneFLOW.

OneFLOW is free software: you can redistribute it and/or modify it
under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

OneFLOW is distributed in the hope that it will be useful, but WITHOUT
ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
for more details.

You should have received a copy of the GNU General Public License
along with OneFLOW.  If not, see <http://www.gnu.org/licenses/>.

\*---------------------------------------------------------------------------*/

#ifdef PRJ_ENABLE_CUDA
#include "Cmpi.h"
#include <omp.h>
#include <cstdio>
#include <cuda_runtime.h>

void SolverInitCuda()
{
    cudaGetDeviceCount( &Cmpi::num_gpus );
    if ( Cmpi::num_gpus < 1 ) {
        std::printf("no CUDA capable devices were detected\n");
        //std::exit(1);
    }

    std::printf("number of host CPUs:\t%d\n", omp_get_num_procs());
    std::printf("number of CUDA devices:\t%d\n", Cmpi::num_gpus);

    for ( int i = 0; i < Cmpi::num_gpus; ++ i )
    {
        cudaDeviceProp dprop;
        cudaGetDeviceProperties( &dprop, i);
        std::printf("   %d: %s\n", i, dprop.name);
    }

    std::printf("---------------------------\n");

    int nCpuThreads = 8;
    omp_set_num_threads( nCpuThreads );
#ifdef ENABLE_OPENMP
#pragma omp parallel
#endif
    {
        unsigned int cpu_thread_id = omp_get_thread_num();
        unsigned int num_cpu_threads = omp_get_num_threads();
        //std::printf( "Solver::Solver() CPU thread %d (of %d)\n", cpu_thread_id, num_cpu_threads );
    }
}

void SetDeviceCuda( int cpu_thread_id )
{
    int gpu_id = -1;
    cudaSetDevice( cpu_thread_id % Cmpi::num_gpus );
    cudaGetDevice( &gpu_id );
}

__global__ void GpuCfdCopyVector( float *a, const float *b, int numElements )
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if ( i < numElements )
    {
        a[i] = b[i];
    }
}

__global__ void GpuCfdScalarUpdate( float * q, const float * qn, float c, const float * timestep, const float * ds, int ni )
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if ( i < ni + 1 && i > 0 )
    {
        float cfl = c * timestep[ i ] / ds[ i ];
        q[ i ] = qn[ i ] - cfl * ( qn[ i ] - qn[ i - 1 ] );
    }
}

void CfdCopyVectorCuda( float * a, float * b, int ni )
{
    std::size_t nSize = ni * sizeof(float);

    float * dev_a;
    float * dev_b;
    cudaMalloc((void **)&dev_a, nSize);
    cudaMalloc((void **)&dev_b, nSize);

    cudaMemcpy(dev_a, a, nSize, cudaMemcpyHostToDevice);
    cudaMemcpy(dev_b, b, nSize, cudaMemcpyHostToDevice);

    int block_size = 256;
    int grid_size = ( ni + block_size - 1 ) / block_size;
    dim3 grid_dim( grid_size );
    dim3 block_dim( block_size );  // 256 threads per block

    GpuCfdCopyVector<<<grid_dim, block_dim>>>( dev_a, dev_b, ni );
    cudaDeviceSynchronize();
    cudaMemcpy(a, dev_a, nSize, cudaMemcpyDeviceToHost);
    cudaFree(dev_a);
    cudaFree(dev_b);
}

void CfdScalarUpdateCuda( float * q, float * qn, float c, float * timestep, float * ds, int ni )
{
    float * dev_q;
    float * dev_qn;
    float * dev_timestep;
    float * dev_ds;
    int nElem = ni + 2;
    std::size_t nSize = nElem * sizeof(float);

    cudaMalloc((void **)&dev_qn, nSize);
    cudaMalloc((void **)&dev_q, nSize);
    cudaMalloc((void **)&dev_timestep, nSize);
    cudaMalloc((void **)&dev_ds, nSize);

    cudaMemcpy(dev_qn, qn, nSize, cudaMemcpyHostToDevice);
    cudaMemcpy(dev_q, q, nSize, cudaMemcpyHostToDevice);
    cudaMemcpy(dev_ds, ds, nSize, cudaMemcpyHostToDevice);
    cudaMemcpy(dev_timestep, timestep, nSize, cudaMemcpyHostToDevice);

    int block_size = 256;
    int grid_size = ( nElem + block_size - 1 ) / block_size;
    dim3 grid_dim( grid_size );
    dim3 block_dim( block_size );  // 256 threads per block

    //std::printf("Solver::SolveField CUDA kernel launch with %d blocks of %d threads\n", grid_size, block_size);
    GpuCfdScalarUpdate<<<grid_dim, block_dim>>>(dev_q, dev_qn, c, dev_timestep, dev_ds, ni);
    cudaDeviceSynchronize();
    cudaMemcpy(q, dev_q, nSize, cudaMemcpyDeviceToHost);
    cudaFree(dev_q);
    cudaFree(dev_qn);
    cudaFree(dev_timestep);
    cudaFree(dev_ds);
}
#endif
