#include "hip/hip_runtime.h"
/*---------------------------------------------------------------------------*\
    OneFLOW - LargeScale Multiphysics Scientific Simulation Environment
    Copyright (C) 2017-2024 He Xin and the OneFLOW contributors.
-------------------------------------------------------------------------------
License
    This file is part of OneFLOW.

    OneFLOW is free software: you can redistribute it and/or modify it
    under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    OneFLOW is distributed in the hope that it will be useful, but WITHOUT
    ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
    FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
    for more details.

    You should have received a copy of the GNU General Public License
    along with OneFLOW.  If not, see <http://www.gnu.org/licenses/>.

\*---------------------------------------------------------------------------*/

#include "SolverDevice.h"
#include "HXMath.h"
#ifdef ENABLE_CUDA
#include "Constant.h"
#include <hip/hip_runtime.h>
#include <iostream>

BeginNameSpace( ONEFLOW )

__global__ void SetValueKernel(Real *dev_a, Real *dev_b, int *dev_id );
__global__ void addKernel(int *a, int *b, int *c );

__global__ void SetValueKernel(Real *dev_a, Real *dev_b, int *dev_id )
{
    int iface = threadIdx.x;
    int icell = dev_id[ iface ];
    dev_a[ iface ] = dev_b[ icell ];
}

__global__ void SetFaceValueCudaDevice(Real *fField, Real *cField, int *iField, int nFaces )
{
    int iFace = blockDim.x * blockIdx.x + threadIdx.x;
    if ( iFace < nFaces )
    {
        int iCell = iField[ iFace ];
        fField[ iFace ] = cField[ iCell ];
    }
}

__global__ void MyInvFluxCuda(Real * qf1, Real * qf2, Real * invflux, Real * xfn, Real * yfn, Real * zfn, Real * area, int nFaces )
{
    int iFace = blockDim.x * blockIdx.x + threadIdx.x;
    if ( iFace < nFaces )
    {
        Real vxl = 1.0;
        Real vyl = 0.0;
        Real vzl = 0.0;

        Real vxr = 1.0;
        Real vyr = 0.0;
        Real vzr = 0.0;

        Real q_L = qf1[ iFace ];
        Real q_R = qf2[ iFace ];

        Real vnl  = xfn[ iFace ] * vxl + yfn[ iFace ] * vyl + zfn[ iFace ] * vzl;
        Real vnr  = xfn[ iFace ] * vxr + yfn[ iFace ] * vyr + zfn[ iFace ] * vzr;

        Real eigenL = vnl;
        Real eigenR = vnr;

        //eigenL = half * ( eigenL + ABS( eigenL ) );
        //eigenR = half * ( eigenR - ABS( eigenR ) );
        eigenL = 0.5 * ( eigenL + abs( eigenL ) );
        eigenR = 0.5 * ( eigenR - abs( eigenR ) );

        Real fL = q_L * eigenL;
        Real fR = q_R * eigenR;
        Real fM = fL + fR;

        Real areaM = area[ iFace ];
        invflux[ iFace ] = fM * areaM;
    }
}

__global__ void MyAddF2CFieldCudaDevice(Real * fField, Real * cField, int * lc, int * rc, int nBFaces, int nFaces )
{
    int iFace = blockDim.x * blockIdx.x + threadIdx.x;
    if ( iFace < nBFaces )
    {
        int lc_ = lc[ iFace ];
        Real value = fField[ iFace ];
        //cField[ lc_ ] -= fField[ iFace ];
        //atomicAdd( &cField[ lc_ ], - value );
    }
    else if ( ( iFace >= nBFaces ) && ( iFace < nFaces ) )
    {
        int lc_ = lc[ iFace ];
        int rc_ = rc[ iFace ];

        Real value = fField[ iFace ];
        //cField[ lc_ ] -= fField[ iFace ];
        //cField[ rc_ ] += fField[ iFace ];
        //atomicAdd( &cField[ lc_ ], - value );
        //atomicAdd( &cField[ rc_ ], value );
    }
}

__global__ void MyAddF2CFieldCudaDeviceNoAtomic(Real * fField, Real * cField, int * lc, int * rc, int nBFaces, int nFaces )
{
    int iFace = blockDim.x * blockIdx.x + threadIdx.x;
    if ( iFace < nBFaces )
    {
        int lc_ = lc[ iFace ];
        cField[ lc_ ] -= fField[ iFace ];
    }
    else if ( ( iFace >= nBFaces ) && ( iFace < nFaces ) )
    {
        int lc_ = lc[ iFace ];
        int rc_ = rc[ iFace ];

        cField[ lc_ ] -= fField[ iFace ];
        cField[ rc_ ] += fField[ iFace ];
    }
}

__global__ void MyZoneTimeIntergralCudaDevice(Real * res, Real * vol, Real dt, int nCells)
{
    int iCell = blockDim.x * blockIdx.x + threadIdx.x;
    if ( iCell < nCells )
    {
        Real ovol = 1.0 / vol[ iCell ];
        Real coef = dt * ovol;
        res[ iCell ] *= coef;
    }
}

__global__ void MyZoneUpdateCudaDevice(Real *q, Real *res, int nCells)
{
    int iCell = blockDim.x * blockIdx.x + threadIdx.x;
    if ( iCell < nCells )
    {
        q[ iCell ] += res[ iCell ];
    }
}

void addWithCuda(int *a, int *b, int *c, unsigned int nElems)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

    hipMalloc((void**)&dev_a, nElems * sizeof(int));
    hipMalloc((void**)&dev_b, nElems * sizeof(int));
    hipMalloc((void**)&dev_c, nElems * sizeof(int));

    hipMemcpy(dev_a, a, nElems * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, nElems * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, nElems>>>(dev_a, dev_b, dev_c);

    hipDeviceSynchronize();

    hipMemcpy(c, dev_c, nElems * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}

__global__ void addKernel(int *a, int *b, int *c )
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void addRealKernel(Real *a, Real *b, Real *c )
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void addRealSwapKernel(Real *a, Real *b, int *id, Real *c )
{
    int i = threadIdx.x;
    int j = id[ i ];
    c[i] = a[i] + b[j];
}

__global__ void setRealSwapKernel(Real *a, int *id, Real *c )
{
    int i = threadIdx.x;
    int j = id[ i ];
    c[i] = a[j];
}

__global__ void setRealSwapKernelNew(Real *a, Real *b, int *id  )
{
    int i = threadIdx.x;
    int j = id[ i ];
    a[i] = b[j];
}

__global__ void setRealSwapKernelNew1(Real *a, Real *b, int *id, int nElems )
{
    //int i = threadIdx.x;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if ( i < nElems )
    {
        int j = id[ i ];
        a[i] = b[j];
    }
}

void addRealWithCuda(Real *a, Real *b, Real *c, unsigned int nElems)
{
    Real *dev_a = 0;
    Real *dev_b = 0;
    Real *dev_c = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

    hipMalloc((void**)&dev_a, nElems * sizeof(Real));
    hipMalloc((void**)&dev_b, nElems * sizeof(Real));
    hipMalloc((void**)&dev_c, nElems * sizeof(Real));

    hipMemcpy(dev_a, a, nElems * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, nElems * sizeof(Real), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    addRealKernel<<<1, nElems>>>(dev_a, dev_b, dev_c);

    hipDeviceSynchronize();

    hipMemcpy(c, dev_c, nElems * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}

void addRealSwapWithCuda(Real *a, Real *b, int * id, Real *c, unsigned int nElems)
{
    Real *dev_a = 0;
    Real *dev_b = 0;
    Real *dev_c = 0;
    int * dev_id = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

    hipMalloc((void**)&dev_a, nElems * sizeof(Real));
    hipMalloc((void**)&dev_b, nElems * sizeof(Real));
    hipMalloc((void**)&dev_c, nElems * sizeof(Real));
    hipMalloc((void**)&dev_id, nElems * sizeof(Real));

    hipMemcpy(dev_a, a, nElems * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, nElems * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_id, id, nElems * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    addRealSwapKernel<<<1, nElems>>>(dev_a, dev_b, dev_id, dev_c);

    hipDeviceSynchronize();

    hipMemcpy(c, dev_c, nElems * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_id);
}

void setRealSwapWithCuda(Real *a, int * id, Real *c, unsigned int nElems)
{
    Real *dev_a = 0;
    Real *dev_c = 0;
    int * dev_id = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

    hipMalloc((void**)&dev_a, nElems * sizeof(Real));
    hipMalloc((void**)&dev_c, nElems * sizeof(Real));
    hipMalloc((void**)&dev_id, nElems * sizeof(Real));

    hipMemcpy(dev_a, a, nElems * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_id, id, nElems * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    setRealSwapKernel<<<1, nElems>>>(dev_a, dev_id, dev_c);

    hipDeviceSynchronize();

    hipMemcpy(c, dev_c, nElems * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_id);
}

void setRealSwapWithCudaNew(Real *a, Real *b, int * id,  unsigned int nElems)
{
    Real *dev_a = 0;
    Real *dev_b = 0;
    int * dev_id = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

    hipMalloc((void**)&dev_a, nElems * sizeof(Real));
    hipMalloc((void**)&dev_b, nElems * sizeof(Real));
    hipMalloc((void**)&dev_id, nElems * sizeof(Real));

    hipMemcpy(dev_b, b, nElems * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_id, id, nElems * sizeof(int), hipMemcpyHostToDevice);

    //setRealSwapKernelNew<<<1, nElems>>>(dev_a, dev_b, dev_id);

    int threadsPerBlock = 256;
    int blocksPerGrid =(nElems + threadsPerBlock - 1) / threadsPerBlock;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    setRealSwapKernelNew1<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_id, nElems);

    hipDeviceSynchronize();

    hipMemcpy(a, dev_a, nElems * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_id);
}

void setRealSwapWithCudaNewRealProblem(Real *a, Real *b, int * id, unsigned int nFaces, unsigned int nCells)
{
    Real *dev_a = 0;
    Real *dev_b = 0;
    int * dev_id = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

    hipMalloc((void**)&dev_a, nFaces * sizeof(Real));
    hipMalloc((void**)&dev_b, nCells * sizeof(Real));
    hipMalloc((void**)&dev_id, nFaces * sizeof(Real));

    hipMemcpy(dev_b, b, nCells * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_id, id, nFaces * sizeof(int), hipMemcpyHostToDevice);

    setRealSwapKernelNew<<<1, nFaces>>>(dev_a, dev_b, dev_id);

    hipDeviceSynchronize();

    hipMemcpy(a, dev_a, nFaces * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_id);
}

void SetFaceValueCuda(Real *fField, Real *cField, int *iField, int nFaces, int nTCells)
{
    Real *dev_fField = 0;
    Real *dev_cField = 0;
    int *dev_iField = 0;

    hipSetDevice(0);

    hipMalloc((void**)&dev_fField, nFaces * sizeof(Real));
    hipMalloc((void**)&dev_cField, nTCells * sizeof(Real));
    hipMalloc((void**)&dev_iField, nFaces * sizeof(int));

    hipMemcpy(dev_fField, fField, nFaces * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_cField, cField, nTCells * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_iField, iField, nFaces * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (nFaces + threadsPerBlock - 1) / threadsPerBlock;

    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    SetFaceValueCudaDevice<<<blocksPerGrid, threadsPerBlock>>>(dev_fField, dev_cField, dev_iField, nFaces );

    hipDeviceSynchronize();

    hipMemcpy(fField, dev_fField, nFaces * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_fField);
    hipFree(dev_cField);
    hipFree(dev_iField);
}

void MyCalcInvFluxCuda(Real *qf1, Real *qf2, Real *invflux, Real *xfn, Real *yfn, Real *zfn, Real *area, int nFaces)
{
    Real *dev_qf1 = 0;
    Real *dev_qf2 = 0;
    Real *dev_invflux = 0;
    Real *dev_xfn = 0;
    Real *dev_yfn = 0;
    Real *dev_zfn = 0;
    Real *dev_area = 0;

    hipSetDevice(0);

    hipMalloc((void**)&dev_qf1, nFaces * sizeof(Real));
    hipMalloc((void**)&dev_qf2, nFaces * sizeof(Real));
    hipMalloc((void**)&dev_invflux, nFaces * sizeof(Real));
    hipMalloc((void**)&dev_xfn, nFaces * sizeof(Real));
    hipMalloc((void**)&dev_yfn, nFaces * sizeof(Real));
    hipMalloc((void**)&dev_zfn, nFaces * sizeof(Real));
    hipMalloc((void**)&dev_area, nFaces * sizeof(Real));

    hipMemcpy(dev_qf1, qf1, nFaces * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_qf2, qf2, nFaces * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_invflux, invflux, nFaces * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_xfn, xfn, nFaces * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_yfn, yfn, nFaces * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_zfn, zfn, nFaces * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_area, area, nFaces * sizeof(Real), hipMemcpyHostToDevice);


    int threadsPerBlock = 256;
    int blocksPerGrid = (nFaces + threadsPerBlock - 1) / threadsPerBlock;

    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    MyInvFluxCuda<<<blocksPerGrid, threadsPerBlock>>>(dev_qf1, dev_qf2, dev_invflux, dev_xfn, dev_yfn, dev_zfn, dev_area, nFaces );

    hipDeviceSynchronize();

    hipMemcpy(invflux, dev_invflux, nFaces * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_qf1);
    hipFree(dev_qf2);
    hipFree(dev_invflux);
    hipFree(dev_xfn);
    hipFree(dev_yfn);
    hipFree(dev_zfn);
    hipFree(dev_area);
}

void MyAddF2CFieldCuda(Real *fField, Real *cField, int *lc, int * rc, int nBFaces, int nFaces, int nTCells)
{
    Real *dev_fField = 0;
    Real *dev_cField = 0;
    int *dev_lc = 0;
    int *dev_rc = 0;

    hipSetDevice(0);

    hipMalloc((void**)&dev_fField, nFaces * sizeof(Real));
    hipMalloc((void**)&dev_cField, nTCells * sizeof(Real));
    hipMalloc((void**)&dev_lc, nFaces * sizeof(int));
    hipMalloc((void**)&dev_rc, nFaces * sizeof(int));

    hipMemcpy(dev_fField, fField, nFaces * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_cField, cField, nTCells * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_lc, lc, nFaces * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_rc, rc, nFaces * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (nFaces + threadsPerBlock - 1) / threadsPerBlock;

    //MyAddF2CFieldCudaDevice<<<blocksPerGrid, threadsPerBlock>>>(dev_fField, dev_cField, dev_lc, dev_rc, nBFaces, nFaces );
    MyAddF2CFieldCudaDeviceNoAtomic<<<blocksPerGrid, threadsPerBlock>>>(dev_fField, dev_cField, dev_lc, dev_rc, nBFaces, nFaces );

    hipDeviceSynchronize();

    hipMemcpy(cField, dev_cField, nTCells * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_fField);
    hipFree(dev_cField);
    hipFree(dev_lc);
    hipFree(dev_rc);
}


void MyZoneTimeIntergralCuda(Real *res, Real *vol, Real dt, int nCells)
{
    Real *dev_res = 0;
    Real *dev_vol = 0;

    hipSetDevice(0);

    hipMalloc((void**)&dev_res, nCells * sizeof(Real));
    hipMalloc((void**)&dev_vol, nCells * sizeof(Real));

    hipMemcpy(dev_res, res, nCells * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_vol, vol, nCells * sizeof(Real), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (nCells + threadsPerBlock - 1) / threadsPerBlock;

    MyZoneTimeIntergralCudaDevice<<<blocksPerGrid, threadsPerBlock>>>(dev_res, dev_vol, dt, nCells);

    hipDeviceSynchronize();

    hipMemcpy(res, dev_res, nCells * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_res);
    hipFree(dev_vol);

}

void MyZoneUpdateCuda(Real *q, Real *res, int nCells)
{
    Real *dev_res = 0;
    Real *dev_q = 0;

    hipSetDevice(0);

    hipMalloc((void**)&dev_res, nCells * sizeof(Real));
    hipMalloc((void**)&dev_q, nCells * sizeof(Real));

    hipMemcpy(dev_res, res, nCells * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(dev_q, q, nCells * sizeof(Real), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (nCells + threadsPerBlock - 1) / threadsPerBlock;

    MyZoneUpdateCudaDevice<<<blocksPerGrid, threadsPerBlock>>>(dev_q, dev_res, nCells);

    hipDeviceSynchronize();

    hipMemcpy(q, dev_q, nCells * sizeof(Real), hipMemcpyDeviceToHost);

    hipFree(dev_q);
    hipFree(dev_res);
}

void TestAdd()
{
    const int arraySize = 5;
    int a[arraySize] = { 1, 2, 3, 4, 5 };
    int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    addWithCuda( a, b, c, arraySize);

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    hipDeviceReset();
}

void TestAddReal()
{
    const int arraySize = 5;
    Real a[arraySize] = { 1.1, 2.2, 3.3, 4.4, 5.5 };
    Real b[arraySize] = { 10, 20, 30, 40, 50 };
    Real c[arraySize] = { 0 };

    // Add vectors in parallel.
    addRealWithCuda( a, b, c, arraySize);

    printf("{1.1, 2.2, 3.3, 4.4, 5.5} + {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",
        c[0], c[1], c[2], c[3], c[4]);

    hipDeviceReset();
}

void TestAddRealSwap()
{
    const int arraySize = 5;
    Real a[arraySize] = { 1.1, 2.2, 3.3, 4.4, 5.5 };
    Real b[arraySize] = { 10, 20, 30, 40, 50 };
    int  id[arraySize] = { 4, 3, 2, 1, 0 };
    Real c[arraySize] = { 0 };

    // Add vectors in parallel.
    addRealSwapWithCuda( a, b, id, c, arraySize);
    //51.1,42.2,33.3,24.4,15.5
    printf("{1.1, 2.2, 3.3, 4.4, 5.5} + {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",
        c[0], c[1], c[2], c[3], c[4]);

    hipDeviceReset();
}

void TestVectorAddRealSwap()
{
    const int arraySize = 5;
    std::vector<Real> a = { 1.1, 2.2, 3.3, 4.4, 5.5 };
    std::vector<Real> b = { 10, 20, 30, 40, 50 };
    std::vector<int>  id = { 4, 3, 2, 1, 0 };
    std::vector<Real> c = { 0,0,0,0,0 };

    // Add vectors in parallel.
    addRealSwapWithCuda( &a[0], &b[0], &id[0], &c[0], arraySize);
    //51.1,42.2,33.3,24.4,15.5
    printf("TestVectorAddRealSwap {1.1, 2.2, 3.3, 4.4, 5.5} + {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",
        c[0], c[1], c[2], c[3], c[4]);

    hipDeviceReset();
}

void TestVectorSetRealSwap()
{
    const int arraySize = 5;
    std::vector<Real> a = { 1.1, 2.2, 3.3, 4.4, 5.5 };
    std::vector<int>  id = { 4, 3, 2, 1, 0 };
    std::vector<Real> c = { 0,0,0,0,0 };

    // Add vectors in parallel.
    setRealSwapWithCuda( &a[0], &id[0], &c[0], arraySize);
    printf("TestVectorSetRealSwap  = {%f,%f,%f,%f,%f}\n",
        c[0], c[1], c[2], c[3], c[4]);

    hipDeviceReset();
}

void TestVectorSetRealSwapNew()
{
    const int arraySize = 5;
    std::vector<int>  id = { 4, 3, 2, 1, 0 };
    std::vector<Real> a = { 0,0,0,0,0 };
    std::vector<Real> b = { 1.1, 2.2, 3.3, 4.4, 5.5 };

    // Add vectors in parallel.
    setRealSwapWithCudaNew( &a[0], &b[0], &id[0], arraySize);
    printf("TestVectorSetRealSwapNew a  = {%f,%f,%f,%f,%f}\n",
        a[0], a[1], a[2], a[3], a[4]);

    hipDeviceReset();
}

void SetRealSwapWithCudaNewRealProblemCpu(Real *a, Real *b, int * id, unsigned int nFaces, unsigned int nCells)
{
    for ( int iFace = 0; iFace < nFaces; ++ iFace )
    {
        int j = id[ iFace ];

        a[ iFace ] = b[ j ];
    }
}

//void TestVectorSetRealSwapNewRealProblem()
//{
//    ScalarGrid * grid = ScalarZone::GetGrid();
//    int nFaces = grid->GetNFaces();
//    int nCells = grid->GetNCells();
//
//    //RealField & q   = GetFieldReference< MRField > ( grid, "q" ).AsOneD();
//    //RealField & qf1 = GetFieldReference< MRField > ( grid, "qf1" ).AsOneD();
//    //RealField & qf2 = GetFieldReference< MRField > ( grid, "qf2" ).AsOneD();
//
//    RealField q   = GetFieldReference< MRField > ( grid, "q" ).AsOneD();
//    RealField qf1 = GetFieldReference< MRField > ( grid, "qf1" ).AsOneD();
//    RealField qf2 = GetFieldReference< MRField > ( grid, "qf2" ).AsOneD();
//
//
//    //std::vector<Real> qq =
//
//    setRealSwapWithCudaNewRealProblem( &qf1[0], &q[0], &grid->lc.data[0], nFaces, nCells );
//    printf("TestVectorSetRealSwapNewRealProblem a  = {%f,%f,%f,%f,%f}\n",
//        qf1[0], qf1[1], qf1[2], qf1[3], qf1[4]);
//
//    hipDeviceReset();
//
//    SetRealSwapWithCudaNewRealProblemCpu( &qf1[0], &q[0], &grid->lc.data[0], nFaces, nCells );
//    printf("TestVectorSetRealSwapNewRealProblem a111  = {%f,%f,%f,%f,%f}\n",
//        qf1[0], qf1[1], qf1[2], qf1[3], qf1[4]);
//    hipDeviceReset();
//
//    std::vector<int> id(nFaces);
//    for( int i = 0; i < nFaces; ++ i )
//    {
//        id[i] = i;
//    }
//    std::vector<Real> xx(nFaces, 0);
//    std::vector<Real> yy(nFaces, 1);
//    setRealSwapWithCudaNewRealProblem( &xx[0], &yy[0], &id[0], nFaces, nFaces );
//    printf("TestVectorSetRealSwapNewRealProblem xx  = {%f,%f,%f,%f,%f}\n",
//        xx[0], xx[1], xx[2], xx[3], xx[4]);
//    hipDeviceReset();
//
//    setRealSwapWithCudaNew( &xx[0], &yy[0], &id[0], nFaces);
//    printf("TestVectorSetRealSwapNew xx111  = {%f,%f,%f,%f,%f}\n",
//        xx[0], xx[1], xx[2], xx[3], xx[4]);
//    hipDeviceReset();
//    TestVectorSetRealSwapNew();
//    hipDeviceReset();
//    //int nnn = 6;
//    //int nnn = 10; //ok
//    //int nnn = 2000; //error
//    //int nnn = 1000; 
//    int nnn = 5000; 
//    std::vector<int>  iddd = { 4, 3, 2, 1, 0 };
//    std::vector<Real> a(nnn, 0);
//    std::vector<Real> b = { 1.1, 2.2, 3.3, 4.4, 5.5 };
//    b.resize( nnn );
//    iddd.resize( nnn );
//
//    // Add vectors in parallel.
//    setRealSwapWithCudaNew( &a[0], &b[0], &iddd[0], nnn);
//    printf("TestVectorSetRealSwapNew a1  = {%f,%f,%f,%f,%f}\n",
//        a[0], a[1], a[2], a[3], a[4]);
//    int kkk = 1;
//}

EndNameSpace
#endif